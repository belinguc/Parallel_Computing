#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdbool.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples
#include "MatUtil.h"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

extern "C"
void computeGold(float *reference, float *idata, const unsigned int len);
void GenMatrix(int *mat, const size_t N);
bool CmpArray(const int *l, const int *r, const size_t eleNum);
void ST_APSP(int *mat, const size_t N);
void printArray(const int *l, const size_t eleNum);

////////////////////////////////////////////////////////////////////////////////
//! APSP basic kernel for device functionality
//! @param g_idata  input data in global memory		Matrix
//! @param k 		input data in global memory		Current k
//! @param N  		input data in global memory		Size of the matrix
////////////////////////////////////////////////////////////////////////////////
__global__ void
apspKernel(int *g_idata, int k, int N)
{
	int mX = blockIdx.x*blockDim.x + threadIdx.x;
	int mY = blockIdx.y*blockDim.y + threadIdx.y;

	int i0 = mX*N + mY;
	int i1 = mX*N + k;
	int i2 = k*N + mY;
		
	if(g_idata[i1] != -1 && g_idata[i2] != -1)
     	{ 
  		int sum =  (g_idata[i1] + g_idata[i2]);
         	if (g_idata[i0] == -1 || sum < g_idata[i0])
		     g_idata[i0] = sum;
	}

}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char **argv)
{
	size_t N = atoi(argv[1]);
	unsigned int mem_size = sizeof(int)*N*N;
	int *mat = (int*)malloc(mem_size);
	GenMatrix(mat, N);

	////////////////////////////////////////////////////////////////////////////////
	//! Compute the reference result
	////////////////////////////////////////////////////////////////////////////////
	StopWatchInterface *timerRef = 0;
	sdkCreateTimer(&timerRef);
	
	int *ref = (int*)malloc(mem_size);
	memcpy(ref, mat, mem_size);

	sdkStartTimer(&timerRef);
	ST_APSP(ref, N);
	sdkStopTimer(&timerRef);

	double tseq = sdkGetTimerValue(&timerRef);
	printf("Processing ref time: %f (ms)\n", tseq);
	// printf("Processing ref time: %f (ms)\n", sdkGetTimerValue(&timerRef));
	sdkDeleteTimer(&timerRef);


	////////////////////////////////////////////////////////////////////////////////
	//! Compute the parallel result withe the APSP basic kernel
	////////////////////////////////////////////////////////////////////////////////
	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
	int devID = findCudaDevice(argc, (const char **)argv);

	StopWatchInterface *timer = 0;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);	
	
	// allocate device memory
	int *d_idata;
	checkCudaErrors(hipMalloc((void **) &d_idata, sizeof(int)*N*N));

	// copy host memory to device
	checkCudaErrors(hipMemcpy(d_idata, mat, sizeof(int)*N*N,hipMemcpyHostToDevice));
	
	// setup execution parameters
	int width = N;
	int tileWidth = 8; // 8x8 = 64 threads/block
	int sizeGrid = ceil(width/tileWidth);
	dim3  dimGrid(sizeGrid, sizeGrid, 1);
	dim3  dimBlock(tileWidth, tileWidth, 1);

	// execute the kernel
	for(int k = 0; k < N; k++){	
		apspKernel<<< dimGrid, dimBlock >>>(d_idata, k, N);
	}

	// check if kernel execution generated and error
	getLastCudaError("Kernel execution failed");

	// allocate mem for the result on host side
	int *result = (int *) malloc(mem_size);

	// copy result from device to host
	checkCudaErrors(hipMemcpyAsync(result, d_idata, sizeof(int) * N*N, hipMemcpyDeviceToHost, 0));

	sdkStopTimer(&timer);
	double tp = sdkGetTimerValue(&timer);
	printf("Processing parallel time: %f (ms)\n", tp);
	sdkDeleteTimer(&timer);


	////////////////////////////////////////////////////////////////////////////////
	//! Compute speedup and compare results
	////////////////////////////////////////////////////////////////////////////////
	double speed = tseq/tp;
	printf("Speed = %f \n", speed);

	//compare results
	if(CmpArray(result, ref, N*N))
	{
		printf("Your result is correct.\n");
		bool bTestResult = true;
	}
	else
	{
		printf("Your result is wrong.\n");
		bool bTestResult = false;
	}

	// cleanup memory
	free(ref);
	checkCudaErrors(hipFree(d_idata));

	hipDeviceReset();
	exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
